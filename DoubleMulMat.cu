#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

const int NUM_REPS = 100;

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}



void devicenfo(void)
{
  struct hipDeviceProp_t capabilities;

  hipGetDeviceProperties (&capabilities, 0);
  printf("->CUDA Platform & Capabilities\n");
  printf("Name: %s\n", capabilities.name);
  printf("totalGlobalMem: %.2f MB\n", capabilities.totalGlobalMem/1024.0f/1024.0f);
  printf("sharedMemPerBlock: %.2f KB\n", capabilities.sharedMemPerBlock/1024.0f);
  printf("regsPerBlock (32 bits): %d\n", capabilities.regsPerBlock);
  printf("warpSize: %d\n", capabilities.warpSize);
  printf("memPitch: %.2f KB\n", capabilities.memPitch/1024.0f);
  printf("maxThreadsPerBlock: %d\n", capabilities.maxThreadsPerBlock);
  printf("maxThreadsDim: %d x %d x %d\n", capabilities.maxThreadsDim[0], 
	 capabilities.maxThreadsDim[1], capabilities.maxThreadsDim[2]);
  printf("maxGridSize: %d x %d\n", capabilities.maxGridSize[0], 
	 capabilities.maxGridSize[1]);
  printf("totalConstMem: %.2f KB\n", capabilities.totalConstMem/1024.0f);
  printf("major.minor: %d.%d\n", capabilities.major, capabilities.minor);
  printf("clockRate: %.2f MHz\n", capabilities.clockRate/1024.0f);
  printf("textureAlignment: %d\n", capabilities.textureAlignment);
  printf("deviceOverlap: %d\n", capabilities.deviceOverlap);
  printf("multiProcessorCount: %d\n", capabilities.multiProcessorCount);
}

void checkResults(float *A, float *B, int width) {
   float maxError = 0.0f;
   for (int i = 0; i < width; i++) {
      for (int j = 0; j < width; j++) {
	      int index = i*width + j;
	      maxError = fmax(maxError, fabs(A[index] - B[index] - 3.0f));
		  printf("%f=%f \n", A[index], B[index]);
	  }
   }
   printf("Max error: %f \n", maxError );
}



__global__ void dgemm(float *M, float *N, float *P, int width) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if ((row < width) && (col < width)) {
		float pVal = 0;
		for (int i = 0; i < width; i++) {
		    pVal = pVal + M[row * width + i] * N[col + i*width]; 
		}
		P[row * width + col] = pVal;
	}
}


void dgemm_cpu(float *M, float *N, float *P, int width) {
    for (int row = 0; row < width; row++) 
	   for (int col = 0; col < width; col++) {
	       float pVal = 0;  
		   for (int k = 0; k < width; k++) {
				pVal = pVal + M[row * width + k] * N[col + k*width]; 
		   }
		   P[row * width + col] = pVal;
	    }	   
}

  
int main(int argc, char **argv)
{
   int width = (argc > 1)?atoi (argv[1]) : 256;
   if (width == 0) {
      devicenfo();
      return(0);
  }
  
  const int mem_size = width*width*sizeof(float);
  int blockSize = 32; 
  int numBlocks = (width + blockSize - 1) / blockSize; 

  dim3 dimBlock(blockSize, blockSize, 1);
  dim3 dimGrid(numBlocks, numBlocks, 1);

  float *A_h = (float*)malloc(mem_size);
  float *B_h = (float*)malloc(mem_size);
  float *C_h = (float*)malloc(mem_size);
  float *D_h = (float*)malloc(mem_size);
  
  float *A_d;
  float *B_d;
  float *C_d;
  
  checkCuda( hipMalloc(&A_d, mem_size) );
  checkCuda( hipMalloc(&B_d, mem_size) );
  checkCuda( hipMalloc(&C_d, mem_size) );
    
  for (int j = 0; j < width; j++) {
    for (int i = 0; i < width; i++) {
	  int index = j*width + i;
      A_h[index] = 2;
	  B_h[index] = 2;
	}  
  }	
  
  // host
  dgemm_cpu(A_h, B_h, D_h, width);
  
  // device
  checkCuda( hipMemcpy(A_d, A_h, mem_size, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(B_d, B_h, mem_size, hipMemcpyHostToDevice) );
  
  // events for timing
  hipEvent_t startEvent, stopEvent;
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  float ms;

  printf("%25s", "matrix multiplication");
  
  checkCuda( hipEventRecord(startEvent, 0) );
  
  dgemm<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, width);
  hipDeviceSynchronize(); 	
	 
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  
  checkCuda( hipMemcpy(C_h, C_d, mem_size, hipMemcpyDeviceToHost) );
  
  checkResults(D_h, C_h, width);

error_exit:
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  
  checkCuda( hipFree(A_d) );
  checkCuda( hipFree(B_d) );
  checkCuda( hipFree(C_d) );
  free(C_h);
  free(A_h);
  free(B_h);
  free(D_h);
}