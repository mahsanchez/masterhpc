
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void checkResults(float *A, float *B, int width) {
   float maxError = 0.0f;
   for (int i = 0; i < width; i++) {
      for (int j = 0; j < width; j++) {
	      int index = i*width + j;
		  //printf("A=%f B=%f \n", A[index], B[index]);
	      maxError = fmax(maxError, fabs(A[index] - B[index] - 3.0f));
	  }
   }
   printf("Max error: %f \n", maxError );
   if (maxError != 3.0f) {
      printf("Unsuccessful results\n");
   }
   else {
      printf("Successful results\n");
   }
}



__global__ void dgemm(float *M, float *N, float *P, int width) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if ((row < width) && (col < width)) {
		float pVal = 0;
		for (int i = 0; i < width; i++) {
		    pVal = pVal + M[row * width + i] * N[col + i*width]; 
		}
		P[row * width + col] = pVal;
	}
}


void dgemm_cpu(float *M, float *N, float *P, int width) {
    for (int row = 0; row < width; row++) 
	   for (int col = 0; col < width; col++) {
	       float pVal = 0;  
		   for (int k = 0; k < width; k++) {
				pVal = pVal + M[row * width + k] * N[col + k*width]; 
		   }
		   P[row * width + col] = pVal;
	    }	   
}

  
int main(int argc, char **argv)
{
   int width = (argc > 1)?atoi (argv[1]) : 256;
  
  const int mem_size = width*width*sizeof(float);
  int blockSize = 32; 
  int numBlocks = (width + blockSize - 1) / blockSize; 

  dim3 dimBlock(blockSize, blockSize, 1);
  dim3 dimGrid(numBlocks, numBlocks, 1);

  float *A_h = (float*)malloc(mem_size);
  float *B_h = (float*)malloc(mem_size);
  float *C_h = (float*)malloc(mem_size);
  float *S_h = (float*)malloc(mem_size);
  float *O_h = (float*)malloc(mem_size);
  
  for (int j = 0; j < width; j++) {
    for (int i = 0; i < width; i++) {
	  int index = j*width + i;
      A_h[index] = 2;
	  B_h[index] = 2;
	  C_h[index] = 2;
	}  
  }	
  
  clock_t tStart = clock();
  dgemm_cpu(A_h, B_h, S_h, width);
  dgemm_cpu(S_h, C_h, O_h, width);
  printf("Time taken by Host: %.6fs\n", (double)(clock() - tStart) / CLOCKS_PER_SEC);
  
  float *A_d;
  float *B_d;
  float *C_d;
  float *O_d;
  float *S_d;
  
  checkCuda( hipMalloc(&A_d, mem_size) );
  checkCuda( hipMalloc(&B_d, mem_size) );
  checkCuda( hipMalloc(&C_d, mem_size) );
  checkCuda( hipMalloc(&S_d, mem_size) );
  checkCuda( hipMalloc(&O_d, mem_size) );
  
  checkCuda( hipMemcpy(A_d, A_h, mem_size, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(B_d, B_h, mem_size, hipMemcpyHostToDevice) );
  
  tStart = clock();
  dgemm<<<dimGrid, dimBlock>>>(A_d, B_d, S_d, width);
  hipDeviceSynchronize();

  checkCuda( hipMemcpy(C_d, C_h, mem_size, hipMemcpyHostToDevice) );
  
  dgemm<<<dimGrid, dimBlock>>>(S_d, C_d, O_d, width);
  hipDeviceSynchronize(); 	
  clock_t tEnd = clock();
  
  checkCuda( hipMemcpy(S_h, O_d, mem_size, hipMemcpyDeviceToHost) );
  
  printf("Time taken by GPU: %.6fs\n", (double)(tEnd - tStart) / CLOCKS_PER_SEC);
  
  checkResults(O_h, S_h, width);

error_exit:
  checkCuda( hipFree(A_d) );
  checkCuda( hipFree(B_d) );
  checkCuda( hipFree(C_d) );
  checkCuda( hipFree(O_d) );
  free(C_h);
  free(A_h);
  free(B_h);
  free(S_h);
  free(O_h);
}