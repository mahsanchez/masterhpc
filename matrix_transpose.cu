#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;
const int NUM_REPS = 100;

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void postprocess(const float *ref, const float *res, int n, float ms)
{
  bool passed = true;
  for (int i = 0; i < n; i++)
    if (res[i] != ref[i]) {
      printf("%d %f %f\n", i, res[i], ref[i]);
      printf("%25s\n", "*** FAILED ***");
      passed = false;
      break;
    }
  if (passed)
    printf("%20.2f\n", 2 * n * sizeof(float) * 1e-6 * NUM_REPS / ms );
}


__global__ void imatcopy(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];
     //tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}
  
int main(int argc, char **argv)
{
  const int nx = 1024;
  const int ny = 1024;
  const int mem_size = nx*ny*sizeof(float);

  dim3 dimGrid(nx/TILE_DIM, ny/TILE_DIM, 1);
  dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

  float *h_idata = (float*)malloc(mem_size);
  float *h_tdata = (float*)malloc(mem_size);
  float *gold    = (float*)malloc(mem_size);
  
  float *d_idata, *d_tdata;
  checkCuda( hipMalloc(&d_idata, mem_size) );
  checkCuda( hipMalloc(&d_tdata, mem_size) );

  // check parameters and calculate execution configuration
  if (nx % TILE_DIM || ny % TILE_DIM) {
    printf("nx and ny must be a multiple of TILE_DIM\n");
    goto error_exit;
  }

  if (TILE_DIM % BLOCK_ROWS) {
    printf("TILE_DIM must be a multiple of BLOCK_ROWS\n");
    goto error_exit;
  }
    
  // host
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < nx; i++)
      h_idata[j*nx + i] = j*nx + i;
  
  // correct result for error checking
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < nx; i++)
      gold[j*nx + i] = h_idata[i*nx + j];
  
  // device
  checkCuda( hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice) );
  
  // events for timing
  hipEvent_t startEvent, stopEvent;
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  float ms;

  printf("%25s", "coalesced transpose");
  checkCuda( hipMemset(d_tdata, 0, mem_size) );
  // warmup
  imatcopy<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  checkCuda( hipEventRecord(startEvent, 0) );
  
  for (int i = 0; i < NUM_REPS; i++)
     imatcopy<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
 
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost) );
  postprocess(gold, h_tdata, nx * ny, ms);

error_exit:
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipFree(d_tdata) );
  checkCuda( hipFree(d_idata) );
  free(h_idata);
  free(h_tdata);
  free(gold);
}