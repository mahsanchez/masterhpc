
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

/*
https://devblogs.nvidia.com/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
https://stackoverflow.com/questions/10460742/how-do-cuda-blocks-warps-threads-map-onto-cuda-cores
*/

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
  }
  return result;
}

__global__ void square(float *array, int k, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  
  for (int i = 0; i < k; i++) {
     int idx = index + i*stride;
     if (idx < n) {
	    array[index + i*stride] = sqrt(array[index + i*stride]);
		//printf("blockIdx.x=%d , blockDim.x=%d , ti=%d, index=%d, i=%d\n", blockIdx.x, blockDim.x, threadIdx.x, index, idx);
	 }
  }
}

void square_cpu(float *array, int n) {
   for (int i = 0; i < n; i++) {
       array[i] = sqrt(array[i]);
   }
}

void initArray(float *a, int size){
  int i;
  for(i=0; i<size; i++){
    a[i] = i*0.01;
  }
}

int main(int argc, char**argv) {
  int N = atoi(argv[1]);
  int threadsPerBlock = atoi(argv[2]);
  int k = atoi(argv[3]);
  
  if ((threadsPerBlock % 32 != 0) || (threadsPerBlock > 1024)) {
     printf("threadsPerBlock must be multiple of 32 and less than 1024");
	 exit(1);
  }
  
  int size = N * sizeof(float);
  float *x = (float*) malloc(size);
  float *y = (float*) malloc(size);
  initArray(x, N);
  memcpy(y, x, size);
  
  clock_t tStart = clock();
  square_cpu(y, N);
  printf("Time taken by Host: %.6fs\n", (double)(clock() - tStart) / CLOCKS_PER_SEC);
  
  float *xd;
  checkCuda( hipMalloc(&xd, size) );
 
  checkCuda( hipMemcpy(xd, x, size, hipMemcpyHostToDevice) );
  
  // Call square kernel
  int blocksPerGrid = (N/k + threadsPerBlock - 1)/threadsPerBlock; 
  
  tStart = clock();
  square<<<blocksPerGrid, threadsPerBlock>>>(xd, k, N);
  hipDeviceSynchronize();
  printf("Time taken by GPU: %.6fs\n", (double)(clock() - tStart) / CLOCKS_PER_SEC);

  checkCuda( hipMemcpy(x, xd, size, hipMemcpyDeviceToHost) );

  // Error Checking
  for (int i = 0; i < N; i++) {
	  if (x[i] != y[i]) {
		  printf("%d %f %f INVALID RESULTS \n", i, x[i], y[i]);
		  goto finalize;
      }
  }	
  
  printf("Successfull Sum\n");

finalize:
  free(x);
  free(y);
  checkCuda(hipFree(xd));
  return 0;
}
