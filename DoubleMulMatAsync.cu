
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void checkResults(float *A, float *B, int width) {
   float maxError = 0.0f;
   for (int i = 0; i < width; i++) {
      for (int j = 0; j < width; j++) {
	      int index = i*width + j;
		  //printf("A=%f B=%f \n", A[index], B[index]);
	      maxError = fmax(maxError, fabs(A[index] - B[index] - 3.0f));
	  }
   }
   printf("Max error: %f \n", maxError );
   if (maxError != 3.0f) {
      printf("Unsuccessful results\n");
   }
   else {
      printf("Successful results\n");
   }
}



__global__ void dgemm(float *M, float *N, float *P, int width) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if ((row < width) && (col < width)) {
		float pVal = 0;
		for (int i = 0; i < width; i++) {
		    pVal = pVal + M[row * width + i] * N[col + i*width]; 
		}
		P[row * width + col] = pVal;
	}
}


void dgemm_cpu(float *M, float *N, float *P, int width) {
    for (int row = 0; row < width; row++) 
	   for (int col = 0; col < width; col++) {
	       float pVal = 0;  
		   for (int k = 0; k < width; k++) {
				pVal = pVal + M[row * width + k] * N[col + k*width]; 
		   }
		   P[row * width + col] = pVal;
	    }	   
}

  
int main(int argc, char **argv)
{
   int width = (argc > 1)?atoi (argv[1]) : 256;
  
  const int mem_size = width*width*sizeof(float);
  int blockSize = 32; 
  int numBlocks = (width + blockSize - 1) / blockSize; 

  dim3 dimBlock(blockSize, blockSize, 1);
  dim3 dimGrid(numBlocks, numBlocks, 1);

  float *A_h;
  float *B_h;
  float *C_h;
  float *S_h;
  float *O_h;
  
  hipHostMalloc(&A_h, mem_size, hipHostMallocDefault);
  hipHostMalloc(&B_h, mem_size, hipHostMallocDefault);
  hipHostMalloc(&C_h, mem_size, hipHostMallocDefault);
  hipHostMalloc(&S_h, mem_size, hipHostMallocDefault);
  hipHostMalloc(&O_h, mem_size, hipHostMallocDefault);
    
  for (int j = 0; j < width; j++) {
    for (int i = 0; i < width; i++) {
	  int index = j*width + i;
      A_h[index] = 2;
	  B_h[index] = 2;
	  C_h[index] = 2;
	}  
  }	
  
  clock_t tStart = clock();
  dgemm_cpu(A_h, B_h, S_h, width);
  dgemm_cpu(S_h, C_h, O_h, width);
  printf("Time taken by Host: %.6fs\n", (double)(clock() - tStart) / CLOCKS_PER_SEC);
  
  float *A_d;
  float *B_d;
  float *C_d;
  float *O_d;
  float *S_d;
  
  checkCuda( hipMalloc(&A_d, mem_size) );
  checkCuda( hipMalloc(&B_d, mem_size) );
  checkCuda( hipMalloc(&C_d, mem_size) );
  checkCuda( hipMalloc(&S_d, mem_size) );
  checkCuda( hipMalloc(&O_d, mem_size) );
  
  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1); 
  hipStreamCreate(&stream2);
  
  checkCuda( hipMemcpyAsync(A_d, A_h, mem_size, hipMemcpyHostToDevice, stream1) );
  checkCuda( hipMemcpyAsync(B_d, B_h, mem_size, hipMemcpyHostToDevice, stream1) );
  
  tStart = clock();
  
  dgemm<<<dimGrid, dimBlock, 0, stream1>>>(A_d, B_d, S_d, width);
  checkCuda( hipMemcpyAsync(C_d, C_h, mem_size, hipMemcpyHostToDevice, stream2) );
  hipDeviceSynchronize();   
  
  dgemm<<<dimGrid, dimBlock, 0, stream2>>>(S_d, C_d, O_d, width);
  hipDeviceSynchronize();  
  clock_t tEnd = clock();
  
  checkCuda( hipMemcpy(S_h, O_d, mem_size, hipMemcpyDeviceToHost) );
  
  printf("Time taken by GPU: %.6fs\n", (double)(tEnd - tStart) / CLOCKS_PER_SEC);
  
  checkResults(O_h, S_h, width);

error_exit:
  hipStreamDestroy(stream1); 
  hipStreamDestroy(stream2);
  checkCuda( hipFree(A_d) );
  checkCuda( hipFree(B_d) );
  checkCuda( hipFree(C_d) );
  checkCuda( hipFree(O_d) );
  checkCuda( hipHostFree(C_h) );
  checkCuda( hipHostFree(A_h) );
  checkCuda( hipHostFree(B_h) );
  checkCuda( hipHostFree(S_h) );
  checkCuda( hipHostFree(O_h) );
}